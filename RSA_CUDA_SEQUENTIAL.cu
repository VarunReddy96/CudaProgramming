
#include <hip/hip_runtime.h>
#include <iostream>


__managed__ unsigned int messagenum = 0;

using namespace std;

// kernal function takes in arguments cipher c, modulus n, messagelist(Which is shared betweeen host
// and device in Unified memory). Because the number of threads and number of blocks in the grid are
// 1 this task becomes similar to sequential.

__global__
void breakingrsa(unsigned long long ciphertext,unsigned long long int n,unsigned long long *messagelist, int ciphermessagescount){
    unsigned long long int thread_pos = blockIdx.x * blockDim.x +threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;
    for (unsigned long long i = thread_pos; i < n; i+=total_threads) {
        unsigned long long val =1 ;
        for(int j =0;j<3;j++) {
            val = ((val * i) % n);
        }
        if(val==ciphertext && messagenum < ciphermessagescount){
            messagelist[atomicAdd(&messagenum, 1)] = i; // Here atomicAdd is used to perform the addition of the RSA messages which are decoded.
                                                        // This  is done by performing atomic addition i,e either it increases the values in the
                                                        // mentioned address or it doesnt and prevents the value from interleaving.
        }
    }
}

// This is is used to sort the messages received in our array messagelist.

void sort(unsigned long long int *messagelist){
    for(int i = 0; i< messagenum;i++){
        for(int j=i+1;j<messagenum;j++){
            if(messagelist[i] > messagelist[j]){
                unsigned long long int tmp = messagelist[i];
                messagelist[i] = messagelist[j];
                messagelist[j] = tmp;
            }
        }
    }

}

// This is the main function takes in aruments the ciphertext c and the modulus n.

int main(int argc, char **argv) {

    if (argc < 3) {
        // When the input arguments are wrong.
        std::cerr << "USAGE: " << argv[0] << " ciphertext" << " " <<argv[1]<< " modulus" << argv[2] << std::endl;
        return 1;
    }

    unsigned long long modulus = stoull(argv[2]);
    unsigned long long cipher = stoull(argv[1]);
    unsigned long long int *messagelist;

    int ciphermessagescount = 100;
    hipMallocManaged(&messagelist,ciphermessagescount*sizeof(unsigned long long));

    breakingrsa<<< 1,1 >>>(cipher,modulus,messagelist,100);

    hipDeviceSynchronize();

    if (messagenum == 0){
        cout << "No cube roots of " << cipher << " (mod " << argv[2] << ")";
    }else{
        sort(messagelist);
        for(int i = 0; i <messagenum;i++){
            cout << messagelist[i] <<"^3 = " << cipher << " (mod " << argv[2] << ")" << endl;
        }
    }

    hipFree(messagelist);
    return 0;
}

